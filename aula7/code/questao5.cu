#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void initializeVector(int *vec, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        vec[idx] = idx;
    }
}
__global__ void squareVector(int *vec, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        vec[idx] = vec[idx] * vec[idx];
    }
}

bool verifyResults(int *vec, int size) {
    for (int i = 0; i < size; i++) {
        if (vec[i] != i * i) {
            printf("Erro na posição %d: Esperado %d, Obtido %d\n", i, i*i, vec[i]);
            return false;
        }
    }
    return true;
}

int main() {
    const int N = 1000;
    int *vec;
    size_t size = N * sizeof(int);
    hipError_t err;

    err = hipMallocManaged(&vec, size);
    if (err != hipSuccess) {
        printf("Erro na alocação: %s\n", hipGetErrorString(err));
        return 1;
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = 32;

    initializeVector<<<blocksPerGrid, threadsPerBlock>>>(vec, N);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Erro na inicialização: %s\n", hipGetErrorString(err));
        hipFree(vec);
        return 1;
    }

    squareVector<<<blocksPerGrid, threadsPerBlock>>>(vec, N);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Erro na operação de quadrado: %s\n", hipGetErrorString(err));
        hipFree(vec);
        return 1;
    }

    if (verifyResults(vec, N)) {
        printf("Sucesso! Todos os elementos foram processados corretamente.\n");
    } else {
        printf("Falha! Erros encontrados nos resultados.\n");
    }

    hipFree(vec);
    return 0;
} 