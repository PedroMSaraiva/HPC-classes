#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void initializeConstant(int *vec, int size, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        vec[idx] = value;
    }
}

__global__ void initializeIncremental(int *vec, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        vec[idx] = idx;
    }
}

__global__ void multiplyVectors(int *vecA, int *vecB, int *result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = vecA[idx] * vecB[idx];
    }
}

int main() {
    const int N = 70;
    int *vecA, *vecB, *result;
    size_t size = N * sizeof(int);

    hipMallocManaged(&vecA, size);
    hipMallocManaged(&vecB, size);
    hipMallocManaged(&result, size);

    int threadsPerBlock = 32;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    initializeConstant<<<blocksPerGrid, threadsPerBlock>>>(vecA, N, 10);
    initializeIncremental<<<blocksPerGrid, threadsPerBlock>>>(vecB, N);
    hipDeviceSynchronize();

    multiplyVectors<<<blocksPerGrid, threadsPerBlock>>>(vecA, vecB, result, N);
    hipDeviceSynchronize();

    printf("Multiplicação dos vetores (elemento a elemento):\n");
    for (int i = 0; i < N; i++) {
        printf("%d * %d = %d\n", vecA[i], vecB[i], result[i]);
    }

    hipFree(vecA);
    hipFree(vecB);
    hipFree(result);

    return 0;
} 